#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "rayData.cuh"
#include "cameraCommon.cuh"

using namespace optix;

rtDeclareVariable(uint2, launchID, rtLaunchIndex, );
rtBuffer<float4, 2> frameResultBuffer;

RT_PROGRAM void pinholeCamera()
{
    RadianceRayData prd;
    prd.result = make_float3(0);
    prd.importance = 1;

    trace(prd, launchID, frameResultBuffer.size());

    frameResultBuffer[launchID] = make_float4(prd.result, 1);
}

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(RadianceRayData, resultRadiance, rtPayload, );
rtDeclareVariable(float3, skyIntensity, , );
rtDeclareVariable(float3, groundIntensity, , );
rtDeclareVariable(float, lightIntensity, , );
rtDeclareVariable(float3, lightColor, , );
rtDeclareVariable(float3, lightDirection, , );
rtDeclareVariable(int, cubemapID, , );

RT_PROGRAM void miss()
{
    float3 direction = normalize(ray.direction);
    float3 normalizedLightDirection = normalize(lightDirection);

    float cosLightAngle = dot(-normalizedLightDirection, direction);
    float3 currentLight = make_float3(0);

    if (cosLightAngle > 0.99998930414f) // cos(0.53 / 180 * pi / 2)
    {
        currentLight = lightColor * lightIntensity;
    }
    else
    {
        float t = clamp((direction.y + 0.5f) / 1.5f, 0.f, 1.f);
        currentLight = lerp(groundIntensity, skyIntensity, t);
    }

    resultRadiance.result = currentLight;
    resultRadiance.importance = 0;
}

RT_PROGRAM void missWithCubemap()
{
    float3 direction = normalize(ray.direction);
    float3 currentLight = make_float3(1.0, 1.0, 1.0);

    if (cubemapID != RT_TEXTURE_ID_NULL)
    {
        //currentLight = make_float3(rtTexCubemap<float4>(cubemapID, ray.direction.x, ray.direction.y, ray.direction.z));
        currentLight = make_float3(rtTexCubemap<float4>(cubemapID, direction.x, direction.y, direction.z));
    }

    resultRadiance.result = currentLight;
    resultRadiance.importance = 0;
}
