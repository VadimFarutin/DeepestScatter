#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "rayData.cuh"

using namespace optix;

rtDeclareVariable(uint2, launchID, rtLaunchIndex, );
rtBuffer<float4, 2>   progressiveBuffer;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtDeclareVariable(rtObject, objectRoot, , );
rtDeclareVariable(float3, errorColor, , );
rtDeclareVariable(float3, missColor, , );

rtDeclareVariable(float, sceneEPS, , );
rtDeclareVariable(unsigned int, radianceRayType, , );
rtDeclareVariable(unsigned int, subframeId, , );

RT_PROGRAM void pinholeCamera()
{
    size_t2 screen = progressiveBuffer.size();

    float2 d = make_float2(launchID) / make_float2(screen) * 2.f - 1.f;

    float3 origin = eye;
    float3 direction = normalize(d.x*U + d.y * V + W);

    optix::Ray ray(origin, direction, radianceRayType, sceneEPS);

    PerRayData_radiance prd;
    prd.importance = 1.0f;
    prd.depth = 0;

    rtTrace(objectRoot, ray, prd); 

    float newWeight = 1.0f / (float)subframeId;
    float oldWeight = 1.0f - newWeight;

    float4 newResult = make_float4(prd.result, 1);

    progressiveBuffer[launchID] = oldWeight * progressiveBuffer[launchID] + newWeight * newResult;
}

RT_PROGRAM void clearScreen()
{
    progressiveBuffer[launchID] = make_float4(0, 0, 0, 1);
}

RT_PROGRAM void exception()
{
    progressiveBuffer[launchID] = make_float4(errorColor, 1);
}

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, resultRadiance, rtPayload, );
rtDeclareVariable(float3, skyIntensity, , );
rtDeclareVariable(float3, groundIntensity, , );
rtDeclareVariable(float, lightIntensity, , );
rtDeclareVariable(float3, lightColor, , );
rtDeclareVariable(float3, lightDirection, , );

RT_PROGRAM void miss()
{
    float3 direction = normalize(ray.direction);
    float3 normalizedLightDirection = normalize(lightDirection);

    float cosLightAngle = dot(-normalizedLightDirection, direction);
    float3 currentLight = make_float3(0);

    if (cosLightAngle > 0.99998930414f) // cos(0.53 / 180 * pi / 2)
    {
        currentLight = lightColor * lightIntensity;
    }
    else
    {
        float t = clamp((direction.y + 0.5f) / 1.5f, 0.f, 1.f);
        currentLight = lerp(groundIntensity, skyIntensity, t);
    }

    resultRadiance.result = currentLight;
    resultRadiance.importance = 0;
}