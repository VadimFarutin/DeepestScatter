#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "rayData.cuh"

using namespace optix;

rtDeclareVariable(uint2, launchID, rtLaunchIndex, );
rtBuffer<float4, 2>   progressiveBuffer;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtDeclareVariable(rtObject, objectRoot, , );
rtDeclareVariable(float3, errorColor, , );
rtDeclareVariable(float3, missColor, , );

rtDeclareVariable(float, sceneEPS, , );
rtDeclareVariable(unsigned int, radianceRayType, , );
rtDeclareVariable(unsigned int, subframeId, , );

RT_PROGRAM void pinholeCamera()
{
    size_t2 screen = progressiveBuffer.size();

    float2 d = make_float2(launchID) / make_float2(screen) * 2.f - 1.f;

    float3 origin = eye;
    float3 direction = normalize(d.x*U + d.y * V + W);

    optix::Ray ray(origin, direction, radianceRayType, sceneEPS);

    PerRayData_radiance prd;
    prd.importance = 1.0f;
    prd.depth = 0;

    rtTrace(objectRoot, ray, prd);

    float newWeight = 1.0f / (float)subframeId;
    float oldWeight = 1.0f - newWeight;

    float4 newResult = make_float4(prd.result, 1);

    progressiveBuffer[launchID] = oldWeight * progressiveBuffer[launchID] + newWeight * newResult;
}

RT_PROGRAM void clearScreen()
{
    progressiveBuffer[launchID] = make_float4(0, 0, 0, 1);
}

RT_PROGRAM void exception()
{
    progressiveBuffer[launchID] = make_float4(errorColor, 1);
}

rtDeclareVariable(PerRayData_radiance, resultRadiance, rtPayload, );

RT_PROGRAM void miss()
{
    resultRadiance.result = missColor;
    resultRadiance.importance = 0;
}