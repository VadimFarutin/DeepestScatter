#include "hip/hip_runtime.h"
﻿#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "rayData.cuh"
#include "random.cuh"

using namespace optix;

rtDeclareVariable(uint2, launchID, rtLaunchIndex, );
rtBuffer<DeepestScatter::Gpu::DisneyNetworkInput, 2> networkInputBuffer;
rtBuffer<IntersectionInfo, 2> directRadianceBuffer;
rtBuffer<float4, 2> frameResultBuffer;

rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtDeclareVariable(float3, lightDirection, , );

rtDeclareVariable(uint2, rectOrigin, , );

rtDeclareVariable(rtObject, objectRoot, , );

rtDeclareVariable(float, sceneEPS, , );
rtDeclareVariable(unsigned int, subframeId, , );

RT_PROGRAM void pinholeCamera()
{
    size_t2 screen = frameResultBuffer.size();

    float2 d = make_float2(launchID + rectOrigin) / make_float2(screen) * 2.f - 1.f;

    uint32_t seed = tea<3>(subframeId);
    float2 jitter = (make_float2(rnd(seed), rnd(seed)) * 2 - 1) / make_float2(screen); // todo;
    d += jitter;

    float3 origin = eye;
    float3 direction = normalize(d.x*U + d.y * V + W);

    DisneyDescriptorRayData prd;
    prd.descriptor = DeepestScatter::Gpu::DisneyDescriptor();
    prd.intersectionInfo.hasScattered = false;
    prd.intersectionInfo.radiance = make_float3(0);

    optix::Ray ray(origin, direction, prd.rayId, sceneEPS);
    rtTrace(objectRoot, ray, prd);

    float angle = acos(dot(lightDirection, direction));

    networkInputBuffer[launchID].fill(prd.descriptor, angle);
    directRadianceBuffer[launchID] = prd.intersectionInfo;
}

RT_PROGRAM void clearRect()
{
    //todo: proably uncomment these. Right now it jut makes performance worse.
    //networkInputBuffer[launchID].clear();
    //directRadianceBuffer[launchID].radiance = make_float3(0);
    //directRadianceBuffer[launchID].hasScattered = false;
}